#include "hip/hip_runtime.h"
#include "WireCellSigProc/cudaOmnibusSigProc.h"


#ifdef HAVE_CUDA_H

#include <hip/hip_runtime.h>

#include <iostream>
#include <omp.h>

#include <thrust/fill.h>
#include <thrust/sort.h>

#include <hipfft/hipfft.h>

#define THREAD_PER_BLOCK 1024
#define ROW_BLOCK 300

// these macros are really really helpful
#define CUDA_SAFE_CALL( call) {                                              \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "CUDA error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
    } }

#define CUFFT_SAFE_CALL( call) {                                             \
    hipfftResult err = call;                                                  \
    if( HIPFFT_SUCCESS != err) {                                              \
        fprintf(stderr, "CUFFT error in file '%s' in line %i : %02X.\n",     \
                __FILE__, __LINE__, err );                                   \
        exit(EXIT_FAILURE);                                                  \
    } }

#define CHECKLASTERROR   {                                                 \
        hipError_t err = hipGetLastError();                                    \
        if (err != hipSuccess) {                                                \
                fprintf(stderr, "CUDA error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
        } }


using namespace std;

using namespace WireCell;

using namespace WireCell::SigProc;

__global__ void ker_restore_baseline_init(float* data, float* data_temp, int* flag,  int row, int col) {

  int index = threadIdx.x + blockIdx.x * blockDim.x;

  if(index < row*col) {
    data_temp[index] = data[index];
    /*
    if(data[index] != 0.0) {
      data_temp[index] = data[index];
      flag[index] = 1;
    } else {
      data_temp[index] = 0.0;
      flag[index] = 0;
    }
    */ 
  }

}

__global__ void ker_restore_baseline_sort1(float* data, float* median, int row, int col) {

  //int index = threadIdx.x + blockIdx.x * blockDim.x;
  int index = blockIdx.x;

  //thrust::device_ptr<float> data(&(data_ptr[index*col]));
 
  if(index < row) { 
    thrust::sort( thrust::device, data + index*col, data + (index+1)*col );
    //thrust::sort( thrust::device, data, data + col );
    //thrust::reduce( thrust::device, data, data + col );
    //thrust::sort( thrust::seq, &(data[index*col]), &(data[(index+1)*col]) );
    //thrust::sort( &(data[index*col]), &(data[(index+1)*col]) );
    median[index] = data[index*col + col/2];
  }

}

__global__ void ker_restore_baseline_sort2(float* data, float* median, int* flag, int row, int col) {

  //int index = threadIdx.x + blockIdx.x * blockDim.x;
  int index = blockIdx.x;

  if(index < row) { 
    thrust::sort( thrust::device, data + index*col, data + (index+1)*col );
    int len = thrust::count( thrust::device, flag + index*col, flag + (index+1)*col, 1 );
    median[index] = data[index*col + len/2];
  }

}

__global__ void ker_restore_baseline_base(float* data, float* median, int* flag,  int row, int col) {


  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int rowIdx = index / col;

  float baseline = median[rowIdx];

  if(index < row*col) {
    float diff = fabs(data[index] - baseline);

    if(diff < 500) {
      data[index] = diff;
      flag[index] = 1;
    } else {
      data[index] = 501.0;
      flag[index] = 0;
    } 
  }

}


__global__ void ker_restore_baseline_shift(float* data, float* median, int row, int col) {


  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int rowIdx = index / col;

  float baseline = median[rowIdx];

  if(index < row*col) {
    if(data[index] != 0.0) data[index] -= baseline; 
  }

}


__global__ void ker_decon_2D_tightROI(hipfftComplex* data, int row, int col, float* filter) {

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int colIdx = index % col;
  float scale = filter[colIdx]/(float)col;

  if(index < row*col) {
    data[index].x *= scale; 
    data[index].y *= scale; 
  }

}


void cudaOmnibusSigProc::init_cudaOmnibusSigProc_CUDA() {
  
  CUDA_SAFE_CALL( hipMalloc(&m_r_data_D, MAX_ROW_M_R_DATA_DEVICE * MAX_COL_M_R_DATA_DEVICE * sizeof(float)) );
  CUDA_SAFE_CALL( hipMalloc(&signal_D, MAX_ROW_M_R_DATA_DEVICE * MAX_COL_M_R_DATA_DEVICE * sizeof(float)) );
  CUDA_SAFE_CALL( hipMalloc(&temp_signal_D, MAX_ROW_M_R_DATA_DEVICE * MAX_COL_M_R_DATA_DEVICE * sizeof(float)) );

  CUDA_SAFE_CALL( hipMalloc(&m_c_data_D, MAX_ROW_M_R_DATA_DEVICE * MAX_COL_M_R_DATA_DEVICE * sizeof(hipfftComplex)) );
  CUDA_SAFE_CALL( hipMalloc(&m_c_data_temp_D, MAX_ROW_M_R_DATA_DEVICE * MAX_COL_M_R_DATA_DEVICE * sizeof(hipfftComplex)) );
  CUDA_SAFE_CALL( hipMalloc(&roi_filter_D, MAX_COL_M_R_DATA_DEVICE * sizeof(float)) );
  log->debug("cudaOmnibusSigProc::init_cudaOmnibusSigProc_CUDA() : hipMalloc() ");

  signal_dev_ptr = thrust::device_malloc<float>(MAX_ROW_M_R_DATA_DEVICE * MAX_COL_M_R_DATA_DEVICE);
  temp_signal_dev_ptr = thrust::device_malloc<float>(MAX_ROW_M_R_DATA_DEVICE * MAX_COL_M_R_DATA_DEVICE);

  signal_flag_dev_ptr = thrust::device_malloc<int>(MAX_ROW_M_R_DATA_DEVICE * MAX_COL_M_R_DATA_DEVICE);
  temp_signal_flag_dev_ptr = thrust::device_malloc<int>(MAX_ROW_M_R_DATA_DEVICE * MAX_COL_M_R_DATA_DEVICE);

  temp_row_dev_ptr = thrust::device_malloc<float>(MAX_ROW_M_R_DATA_DEVICE);


  for(int i=0 ; i<STREAM_NUM ; i++) {
    CUDA_SAFE_CALL( hipStreamCreate(&m_Streams[i]) );
  }


}

void cudaOmnibusSigProc::upload_c_data(std::complex<float>* c, int row, int col) {

  CUDA_SAFE_CALL( hipMemcpy(m_c_data_D, c, row*col*sizeof(std::complex<float>), hipMemcpyHostToDevice) );

}


void cudaOmnibusSigProc::clean_cudaOmnibusSigProc_CUDA() {
  CUDA_SAFE_CALL(hipFree(m_r_data_D));
  CUDA_SAFE_CALL(hipFree(signal_D));
  CUDA_SAFE_CALL(hipFree(temp_signal_D));

  thrust::device_free(signal_dev_ptr);
  thrust::device_free(temp_signal_dev_ptr);

  thrust::device_free(signal_flag_dev_ptr);
  thrust::device_free(temp_signal_flag_dev_ptr);

  thrust::device_free(temp_row_dev_ptr);

  for(int i=0 ; i<STREAM_NUM ; i++) {
    hipStreamDestroy(m_Streams[i]);
  }

}

void cudaOmnibusSigProc::restore_baseline_CUDA(float* data, int row, int col) {

  if(row % STREAM_NUM != 0) {
    cout << "====================================================================" << endl;
    cout << "restore_base_CUDA() : row=" << row << ", STREAM_NUM=" << STREAM_NUM << endl;
    cout << "====================================================================" << endl;

  }

  int row_block_size = row / STREAM_NUM;
  cout << "restore_base_CUDA() : row_block_size=" << row_block_size << endl;

  

  //CUDA_SAFE_CALL( hipHostRegister(data, row*col*sizeof(float), hipHostRegisterDefault) );
  hipHostRegister(data, row*col*sizeof(float), hipHostRegisterPortable);
  //hipHostRegister(data, MAX_ROW_M_R_DATA_DEVICE*MAX_COL_M_R_DATA_DEVICE*sizeof(float), hipHostRegisterPortable);


  double wstart, wend;


  for(int s=0 ; s<STREAM_NUM ; s++) {

  /*
  cout << "signal : ";
  for(int i=0 ; i<col ; i++) {
    cout << data[i] << ", ";
  }
  cout << endl;
  */

  //CUDA_SAFE_CALL( hipMemcpyAsync(m_r_data_D + s*row_block_size*col, data + s*row_block_size*col, row_block_size*col*sizeof(float), hipMemcpyHostToDevice, m_Streams[s]) );

  //log->debug("cudaOmnibusSigProc::restore_baseline_CUDA() : hipMemcpy() ");

  //ker_restore_baseline_init<<<(row*col)/THREAD_PER_BLOCK + 1, THREAD_PER_BLOCK>>>(m_r_data_D, thrust::raw_pointer_cast(signal_dev_ptr), thrust::raw_pointer_cast(signal_flag_dev_ptr), row, col);
  ker_restore_baseline_init<<<(row_block_size*col)/THREAD_PER_BLOCK + 1, THREAD_PER_BLOCK, 0, m_Streams[s]>>>(m_r_data_D + s*row_block_size*col, thrust::raw_pointer_cast(signal_dev_ptr) + s*row_block_size*col, thrust::raw_pointer_cast(signal_flag_dev_ptr) + s*row_block_size*col, row_block_size, col);

  /*
  int iter = row / ROW_BLOCK;
  (row % ROW_BLOCK == 0) ? : iter++;
  for (int i=0 ; i<iter ; i++) {
    ker_restore_baseline_sort1<<<ROW_BLOCK, 1>>>(thrust::raw_pointer_cast(signal_dev_ptr), thrust::raw_pointer_cast(temp_row_dev_ptr), row, col, i);
  }
  */

  int iter_size = row_block_size / 4;
  for(int i=0 ; i<4 ; i++) {
    ker_restore_baseline_sort1<<<row_block_size, 1, 0, m_Streams[s]>>>(thrust::raw_pointer_cast(signal_dev_ptr) + s*row_block_size*col + i*iter_size*col, thrust::raw_pointer_cast(temp_row_dev_ptr) + s*row_block_size + i*iter_size, iter_size, col);
  }


  /*
  wstart = omp_get_wtime();
  for (int i=0 ; i<row ; i++){
    thrust::sort( &(signal_dev_ptr[i*col]), &(signal_dev_ptr[(i+1)*col]) );
    temp_row_dev_ptr[i] = signal_dev_ptr[i*col + col/2];
    //float median = signal_dev_ptr[i*col + col/2];
    //cout << "[i:"<<i<<"] basseline : " << median << endl;
    //float median = signal_dev_ptr[i*col];
    //cout << "[i:"<<i<<"] basseline : " << median << endl;
  } 
  wend = omp_get_wtime();
  cout << "first sort : " << wend - wstart << endl;
  */


  //ker_restore_baseline_base<<<(row*col)/THREAD_PER_BLOCK + 1, THREAD_PER_BLOCK>>>(thrust::raw_pointer_cast(signal_dev_ptr), thrust::raw_pointer_cast(temp_row_dev_ptr), thrust::raw_pointer_cast(signal_flag_dev_ptr), row, col);
  ker_restore_baseline_base<<<(row_block_size*col)/THREAD_PER_BLOCK + 1, THREAD_PER_BLOCK, 0, m_Streams[s]>>>(thrust::raw_pointer_cast(signal_dev_ptr) + s*row_block_size*col, thrust::raw_pointer_cast(temp_row_dev_ptr) + s*row_block_size, thrust::raw_pointer_cast(signal_flag_dev_ptr) + s*row_block_size*col, row_block_size, col);

  /*
  for (int i=0 ; i<iter ; i++) {
    ker_restore_baseline_sort2<<<ROW_BLOCK, 1>>>(thrust::raw_pointer_cast(signal_dev_ptr), thrust::raw_pointer_cast(temp_row_dev_ptr), thrust::raw_pointer_cast(signal_flag_dev_ptr), row, col, i);
  }
  */

  for(int i=0 ; i<4 ; i++) {
    ker_restore_baseline_sort2<<<row_block_size, 1, 0, m_Streams[s]>>>(thrust::raw_pointer_cast(signal_dev_ptr) + s*row_block_size*col + i*iter_size*col, thrust::raw_pointer_cast(temp_row_dev_ptr) + s*row_block_size + i*iter_size, thrust::raw_pointer_cast(signal_flag_dev_ptr) + s*row_block_size*col + i*iter_size*col, iter_size, col);
  }

  /*
  wstart = omp_get_wtime();
  for (int i=0 ; i<row ; i++){
    thrust::sort( &(signal_dev_ptr[i*col]), &(signal_dev_ptr[(i+1)*col]) );
    int len = thrust::count( &(signal_flag_dev_ptr[i*col]), &(signal_flag_dev_ptr[(i+1)*col]), 1 );

    temp_row_dev_ptr[i] = signal_dev_ptr[i*col + len/2];

  }
  wend = omp_get_wtime();
  cout << "second sort : " << wend - wstart << endl;
  */

  ker_restore_baseline_shift<<<(row_block_size*col)/THREAD_PER_BLOCK + 1, THREAD_PER_BLOCK, 0, m_Streams[s]>>>(m_r_data_D + s*row_block_size*col, thrust::raw_pointer_cast(temp_row_dev_ptr) + s*row_block_size, row_block_size, col);


  CUDA_SAFE_CALL( hipMemcpyAsync(data + s*row_block_size*col, m_r_data_D + s*row_block_size*col, row_block_size*col*sizeof(float), hipMemcpyDeviceToHost, m_Streams[s]) );

  } // for stream

  hipDeviceSynchronize();
  CUDA_SAFE_CALL( hipHostUnregister(data) );
  
}



void cudaOmnibusSigProc::decon_2D_tightROI_CUDA(int row, int col, float* filter, float* r_data, int start_row, int start_col, int row_len, int col_len) {

  
  CUDA_SAFE_CALL( hipMemcpy(roi_filter_D, filter, col*sizeof(float), hipMemcpyHostToDevice) );

  ker_decon_2D_tightROI<<<(row*col)/THREAD_PER_BLOCK + 1, THREAD_PER_BLOCK>>>(m_c_data_D, row, col, roi_filter_D);
  CHECKLASTERROR;
  //hipDeviceSynchronize();

  //std::complex<float>* c_data = (std::complex<float> *)( malloc(row*col*sizeof(std::complex<float>)) );
  //CUDA_SAFE_CALL( hipMemcpy(c_data, m_c_data_D, row*col*sizeof(std::complex<float>), hipMemcpyDeviceToHost) );
  //cout << "cudaOmnibusSigProc::decon_2D_tightROI_CUDA() : real : " << real(c_data[100]) << ", image : " << imag(c_data[100]) << endl;
  //cout << "cudaOmnibusSigProc::decon_2D_tightROI_CUDA() : filter : " << filter[100] << ", row : " << row << ", col : " << col << endl;


  hipfftHandle handle;
  int rank = 1;                   // --- 1D FFTs
  int n[] = { col };              // --- Size of the Fourier transform
  int istride = 1, ostride = 1;   // --- Distance between two successive input/output elements
  int idist = col, odist = col;   // --- Distance between batches
  int inembed[] = { 0 };          // --- Input size with pitch (ignored for 1D transforms)
  int onembed[] = { 0 };          // --- Output size with pitch (ignored for 1D transforms)
  int batch = row;                // --- Number of batched executions
  CUFFT_SAFE_CALL( hipfftPlanMany(&handle, rank, n, 
              inembed, istride, idist,
              onembed, ostride, odist, HIPFFT_C2R, batch) );

  CUFFT_SAFE_CALL( hipfftExecC2R(handle,  m_c_data_D, m_r_data_D) );

  CUFFT_SAFE_CALL( hipfftDestroy(handle) );

  if(!(start_row == 0 && row == row_len && col == col_len)) {
    cout << "====================================================================" << endl;
    cout << "decon_2D_tightROI_CUDA() : start_row :" << start_row << ", row_len : " << row_len << ", col_len : " << col_len << endl;
    cout << "====================================================================" << endl;
  }




  /*
  ker_restore_baseline_init<<<(row*col)/THREAD_PER_BLOCK + 1, THREAD_PER_BLOCK>>>(m_r_data_D, thrust::raw_pointer_cast(signal_dev_ptr), thrust::raw_pointer_cast(signal_flag_dev_ptr), row, col);
  CHECKLASTERROR;

  int iter_size = row / 4;
  for(int i=0 ; i<4 ; i++) {
    ker_restore_baseline_sort1<<<row, 1>>>(thrust::raw_pointer_cast(signal_dev_ptr), thrust::raw_pointer_cast(temp_row_dev_ptr), iter_size, col);
  }

  ker_restore_baseline_base<<<(row*col)/THREAD_PER_BLOCK + 1, THREAD_PER_BLOCK>>>(thrust::raw_pointer_cast(signal_dev_ptr), thrust::raw_pointer_cast(temp_row_dev_ptr), thrust::raw_pointer_cast(signal_flag_dev_ptr), row, col);
  CHECKLASTERROR;

  for(int i=0 ; i<4 ; i++) {
    ker_restore_baseline_sort2<<<row, 1>>>(thrust::raw_pointer_cast(signal_dev_ptr), thrust::raw_pointer_cast(temp_row_dev_ptr), thrust::raw_pointer_cast(signal_flag_dev_ptr), iter_size, col);
  }

  ker_restore_baseline_shift<<<(row*col)/THREAD_PER_BLOCK + 1, THREAD_PER_BLOCK>>>(m_r_data_D, thrust::raw_pointer_cast(temp_row_dev_ptr), row, col);
  CHECKLASTERROR;
  */

  CUDA_SAFE_CALL( hipMemcpy(r_data, m_r_data_D, row*col*sizeof(float), hipMemcpyDeviceToHost) );
  //hipDeviceSynchronize();


}

#endif


